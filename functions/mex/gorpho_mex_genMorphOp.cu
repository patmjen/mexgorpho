#include "hip/hip_runtime.h"
#include <stdexcept>

#include "mex.h"

#include "view.cuh"
#include "general_morph.cuh"

#include "mex_common.cuh"

template <class Ty>
void genMorphOp(mxArray *res, const mxArray *vol, const mxArray *strel, int op, int3 blockSize);

/** Grayscale morphological operation with grayscale structuring element
 *
 * Parameters
 * ----------
 * vol : numerical array
 *     Input volume.
 * strel : numerical array
 *     Structuring element. Must have same class as input volume.
 * op : numerical
 *     Operation to perform.
 * blockSize : numerical vector
 *     Block size for GPU processing.
 *
 * Returns
 * -------
 * result : numerical array
 *     Output volume with result of operation. Same size as input volume.
 */
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
    // Validate inputs
    ensureOrError(nrhs == 4, "Must supply 4 inputs");
    ensureOrError(nlhs == 1, "Must have 1 output");

    const mxArray *mxVol = prhs[0];
    const mxArray *mxStrel = prhs[1];
    ensureRealNumericVolume(mxVol, "vol");
    ensureRealNumericVolume(mxStrel, "strel");
    ensureOrError(mxGetClassID(mxVol) == mxGetClassID(mxStrel), "vol and strel must have same class");
    int op = getValidatedMorphOp(prhs[2]);
    int3 blockSize = getValidatedInt3(prhs[3]);
    ensureOrError(blockSize.x > 0 && blockSize.y > 0 && blockSize.z > 0, "blockSize must be positive");

    // Allocate output
    mxArray *mxRes = createVolumeLike(mxVol);

    // Run function
    typeDispatch(mxGetClassID(prhs[0]), genMorphOp, mxRes, mxVol, mxStrel, op, blockSize);
    plhs[0] = mxRes;
}

template <class Ty>
void genMorphOp(mxArray *mxRes, const mxArray *mxVol, const mxArray *mxStrel, int op, int3 blockSize)
{
    gpho::HostView<Ty> res(static_cast<Ty *>(mxGetData(mxRes)), volSize(mxRes));
    gpho::HostView<const Ty> vol(static_cast<const Ty *>(mxGetData(mxVol)), volSize(mxVol));
    gpho::HostView<const Ty> strel(static_cast<const Ty *>(mxGetData(mxStrel)), volSize(mxStrel));

    switch (op) {
    case MOP_DILATE:
        gpho::genDilate(res, vol, strel, blockSize);
        break;
    case MOP_ERODE:
        gpho::genErode(res, vol, strel, blockSize);
        break;
    default:
        abortWithMsg("invalid morphology op");
    }
}