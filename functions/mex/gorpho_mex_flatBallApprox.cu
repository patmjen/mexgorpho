#include <stdexcept>

#include "mex.h"
#include "matrix.h"

#include "flat_linear_morph.cuh"
#include "strel.cuh"

#include "mex_common.cuh"

inline gpho::ApproxType toApproxType(int approxType)
{
    if (approxType == 0) {
        return gpho::APPROX_INSIDE;
    } else if (approxType == 1) {
        return gpho::APPROX_BEST;
    } else {
        return gpho::APPROX_OUTSIDE;
    }
}

/** Line segment approximation to flat ball structuring element
 *
 * Parameters
 * ----------
 * radius : numeric scalar
 *     Radius of ball.
 * approxType : numeric scalar
 *     Type of approximation: 0 = constrained inside, 1 = best, 2 = constrained outside.
 *
 * Returns
 * -------
 * lineSteps : int32 matrix
 *     N x 3 matrix with step vectors for line segments.
 * lineLens : int32 vector
 *     N x 1 vector with length sof line segments (in steps).
 */
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
    // Validate inputs
    ensureOrError(nrhs == 2, "Must supply 2 inputs");
    ensureOrError(nlhs == 2, "Must have 2 outputs");

    int radius = getValidatedScalar<int>(prhs[0], "radius");
    ensureOrError(radius > 0, "radius mus the positive");
    int approxType = getValidatedScalar<int>(prhs[1], "approxType");
    ensureValue(approxType, { 0, 1, 2 }, "approxType");

    std::vector<gpho::LineSeg> lines = gpho::flatBallApprox(radius, toApproxType(approxType));

    // Allocate and fill outputs
    mxArray *mxLineSteps = mxCreateUninitNumericMatrix(lines.size(), 3, mxINT32_CLASS, mxREAL);
    mxArray *mxLineLens = mxCreateUninitNumericMatrix(lines.size(), 1, mxINT32_CLASS, mxREAL);
    int *lineStepsData = static_cast<int *>(mxGetData(mxLineSteps));
    int *lineLensData = static_cast<int *>(mxGetData(mxLineLens));

    for (size_t i = 0; i < lines.size(); ++i) {
        const auto& ls = lines[i];
        lineStepsData[i + 0 * lines.size()] = ls.step.x;
        lineStepsData[i + 1 * lines.size()] = ls.step.y;
        lineStepsData[i + 2 * lines.size()] = ls.step.z;
        lineLensData[i] = ls.length;
    }

    plhs[0] = mxLineSteps;
    plhs[1] = mxLineLens;
}