#include "hip/hip_runtime.h"
#include <stdexcept>
#include <vector>

#include "mex.h"

#include "view.cuh"
#include "morph.cuh"
#include "strel.cuh"
#include "flat_linear_morph.cuh"

#include "mex_common.cuh"

template <class Ty>
void flatLinearMorphOp(mxArray *mxRes, const mxArray *mxVol, const std::vector<gpho::LineSeg>& lines,
    int op, int3 blockSize);

/** Grayscale morphological operation with flat linear structuring elements
 *
 * Parameters
 * ----------
 * vol : numeric array
 *     Input volume.
 * lineSteps : int32 matrix
 *     Step vectors for line segments. Must be N x 3.
 * lineLens : int32 vector
 *     Number of steps for line segments.
 * op : numeric
 *     Operation to perform.
 * blockSize : numeric vector
 *     Block size for GPU processing.
 *
 * Returns
 * -------
 * result : numeric array
 *     Output volume with result of operation. Same size and class as input volume.
 */
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
    // Validate inputs
    ensureOrError(nrhs == 5, "Must supply 5 inputs");
    ensureOrError(nlhs == 1, "Must have 1 output");

    const mxArray *mxVol = prhs[0];
    const mxArray *mxLineSteps = prhs[1];
    const mxArray *mxLineLens = prhs[2];
    ensureRealNumericVolume(mxVol, "vol");
    ensureRealMatrix(mxLineSteps, "lineSteps");
    ensureOrError(mxGetN(mxLineSteps) == 3, "lineSteps must be N x 3");
    ensureOrError(mxGetClassID(mxLineSteps) == mxINT32_CLASS, "lineSteps must be int32");
    ensureRealVector(mxLineLens, "lineLens");
    ensureOrError(mxGetClassID(mxLineSteps) == mxINT32_CLASS, "lineLens must be int32");
    ensureOrError(mxGetM(mxLineSteps) == mxGetNumberOfElements(mxLineLens),
        "number of steps vectors and step lengths must be equal");
    int op = getValidatedMorphOp(prhs[3]);
    int3 blockSize = getValidatedInt3(prhs[4]);
    ensureOrError(blockSize.x > 0 && blockSize.y > 0 && blockSize.z > 0, "blockSize must be positive");

    // Allocate output
    mxArray *mxRes = createVolumeLike(mxVol);

    // Extract line segments
    size_t numLineSegs = mxGetM(mxLineSteps);
    std::vector<gpho::LineSeg> lines(numLineSegs);
    const int *lineStepsData = static_cast<const int *>(mxGetData(mxLineSteps));
    const int *lineLensData = static_cast<const int *>(mxGetData(mxLineLens));
    for (size_t i = 0; i < numLineSegs; ++i) {
        // MATLAB stores matrices in column major order
        auto& ls = lines[i];
        ls.step.x = lineStepsData[i + 0 * numLineSegs];
        ls.step.y = lineStepsData[i + 1 * numLineSegs];
        ls.step.z = lineStepsData[i + 2 * numLineSegs];
        ls.length = lineLensData[i];
    }

    // Run function
    typeDispatch(mxGetClassID(prhs[0]), flatLinearMorphOp, mxRes, mxVol, lines, op, blockSize);
    plhs[0] = mxRes;
}

template <class Ty>
void flatLinearMorphOp(mxArray *mxRes, const mxArray *mxVol, const std::vector<gpho::LineSeg>& lines,
    int op, int3 blockSize)
{
    gpho::HostView<Ty> res(static_cast<Ty *>(mxGetData(mxRes)), volSize(mxRes));
    gpho::HostView<const Ty> vol(static_cast<const Ty *>(mxGetData(mxVol)), volSize(mxVol));

    switch (op) {
    case MOP_DILATE:
        gpho::flatLinearDilateErode<gpho::MORPH_DILATE>(res, vol, lines, blockSize);
        break;
    case MOP_ERODE:
        gpho::flatLinearDilateErode<gpho::MORPH_ERODE>(res, vol, lines, blockSize);
        break;
    default:
        abortWithMsg("invalid morphology op");
    }
}