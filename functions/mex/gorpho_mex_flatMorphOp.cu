#include "hip/hip_runtime.h"
#include <stdexcept>

#include "mex.h"

#include "view.cuh"
#include "flat_morph.cuh"

#include "mex_common.cuh"

template <class Ty>
void flatMorphOp(mxArray *res, const mxArray *vol, const mxArray *strel, int op, int3 blockSize);

/** Grayscale morphological operation with flat structuring element
 *
 * Parameters
 * ----------
 * vol : numerical array
 *     Input volume.
 * strel : logical array
 *     Structuring element.
 * op : numerical
 *     Operation to perform.
 * blockSize : numerical vector
 *     Block size for GPU processing.
 *
 * Returns
 * -------
 * result : numerical array
 *     Output volume with result of operation. Same size as input volume.
 */
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
    // Validate inputs
    ensureOrError(nrhs == 4, "Must supply 4 inputs");
    ensureOrError(nlhs == 1, "Must have 1 output");

    const mxArray *mxVol = prhs[0];
    const mxArray *mxStrel = prhs[1];
    ensureRealNumericVolume(mxVol, "vol");
    ensureOrError(isRealVolume(mxStrel) && mxIsLogical(mxStrel), "strel must a real logical volume");
    int op = getValidatedMorphOp(prhs[2]);
    int3 blockSize = getValidatedInt3(prhs[3]);
    ensureOrError(blockSize.x > 0 && blockSize.y > 0 && blockSize.z > 0, "blockSize must be positive");

    // Allocate output
    mxArray *mxRes = createVolumeLike(mxVol);

    // Run function
    typeDispatch(mxGetClassID(prhs[0]), flatMorphOp, mxRes, mxVol, mxStrel, op, blockSize);
    plhs[0] = mxRes;
}

template <class Ty>
void flatMorphOp(mxArray *mxRes, const mxArray *mxVol, const mxArray *mxStrel, int op, int3 blockSize)
{
    gpho::HostView<Ty> res(static_cast<Ty *>(mxGetData(mxRes)), volSize(mxRes));
    gpho::HostView<const Ty> vol(static_cast<const Ty *>(mxGetData(mxVol)), volSize(mxVol));
    gpho::HostView<const bool> strel(static_cast<const bool *>(mxGetData(mxStrel)), volSize(mxStrel));

    switch (op) {
    case MOP_DILATE:
        gpho::flatDilate(res, vol, strel, blockSize);
        break;
    case MOP_ERODE:
        gpho::flatErode(res, vol, strel, blockSize);
        break;
    case MOP_OPEN:
        gpho::flatOpen(res, vol, strel, blockSize);
        break;
    case MOP_CLOSE:
        gpho::flatClose(res, vol, strel, blockSize);
        break;
    case MOP_TOPHAT:
        gpho::flatTophat(res, vol, strel, blockSize);
        break;
    case MOP_BOTHAT:
        gpho::flatBothat(res, vol, strel, blockSize);
        break;
    default:
        abortWithMsg("invalid morphology op");
    }
}